#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************
  This program gives an example of a poor way to implement a password cracker
  in CUDA C. It is poor because it acheives this with just one thread, which
  is obviously not good given the scale of parallelism available to CUDA
  programs.
  
  The intentions of this program are:
    1) Demonstrate the use of __device__ and __global__ functions
    2) Enable a simulation of password cracking in the absence of library 
       with equivalent functionality to libcrypt. The password to be found
       is hardcoded into a function called is_a_match.   

  Compile and run with:
    nvcc -o CrackAZ99Cuda CrackAZ99Cuda.cu
    ./CrackAZ99Cuda
   
  Dr Kevan Buckley, University of Wolverhampton, 2018
*****************************************************************************/

/****************************************************************************
  This function returns 1 if the attempt at cracking the password is 
  identical to the plain text password string stored in the program. 
  Otherwise,it returns 0.
*****************************************************************************/

__device__ int is_a_match(char *attempt) {
	char mypassword1[] = "SA1234";
	char mypassword2[] = "MI4567";
	char mypassword3[] = "RC7890";
	char mypassword4[] = "HA2345";


	char *p = attempt;
	char *r = attempt;
	char *a = attempt;
	char *t = attempt;
	char *p1 = mypassword1;
	char *p2 = mypassword2;
	char *p3 = mypassword3;
	char *p4 = mypassword4;

	while(*p == *p1) { 
		if(*p == '\0') 
		{
			printf("Password: %s\n",mypassword1);
			break;
		}

		p++;
		p1++;
	}
	
	while(*r == *p2) { 
		if(*r == '\0') 
		{
			printf("Password: %s\n",mypassword2);
			break;
		}

		r++;
		p2++;
	}

	while(*a == *p3) { 
		if(*a == '\0') 
		{
			printf("Password: %s\n",mypassword3);
			break;
		}

		a++;
		p3++;
	}

	while(*t == *p4) { 
		if(*t == '\0') 
		{
			printf("Password: %s\n",mypassword4);
			return 1;
		}

		t++;
		p4++;
	}
	return 0;

}

__global__ void  kernel() {
	char i1,i2,i3,i4;

	char password[7];
	password[6] = '\0';

	int i = blockIdx.x+65;
	int j = threadIdx.x+65;
	char firstMatch = i; 
	char secondMatch = j; 

	password[0] = firstMatch;
	password[1] = secondMatch;
	for(i1='0'; i1<='9'; i1++){
		for(i2='0'; i2<='9'; i2++){
			for(i3='0'; i3<='9'; i3++){
				for(i4='0'; i4<='9'; i4++){
					password[2] = i1;
					password[3] = i2;
					password[4] = i3;
					password[5] = i4; 
					if(is_a_match(password)) {
					} 
				}
			}
		}
	}
}

int time_difference(struct timespec *start, 
	struct timespec *finish, 
	long long int *difference) {
	long long int ds =  finish->tv_sec - start->tv_sec; 
	long long int dn =  finish->tv_nsec - start->tv_nsec; 
	if(dn < 0 ) {
		ds--;
		dn += 1000000000; 
	} 
	*difference = ds * 1000000000 + dn;
	return !(*difference > 0);
}


int main() {

	struct  timespec start, finish;
	long long int time_elapsed;
	clock_gettime(CLOCK_MONOTONIC, &start);

	kernel <<<26,26>>>();
	hipDeviceSynchronize();

	clock_gettime(CLOCK_MONOTONIC, &finish);
	time_difference(&start, &finish, &time_elapsed);
	printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9)); 

	return 0;
}


